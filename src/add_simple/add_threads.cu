/*
 * Code originally from https://devblogs.nvidia.com/even-easier-introduction-cuda/
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// function to add the elements of two arrays
__global__ // this makes it run on the gpu
void add(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
      y[i] = x[i] + y[i];
}

__global__
void add_threads(int n, float*x, float*y)
{
	int index = threadIdx.x;
	int stride = blockDim.x;
	for (int i=index; i<n; i+=stride)
	{
		y[i] = x[i] + y[i];
	}
}

int main(void)
{
  int N = 1<<20; // 1M elements

  //float *x = new float[N];
  //float *y = new float[N];
  // Allocating "Unified Memory" -- can be accessed from both cpu and gpu
  float *x, *y;
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the CPU
  //add(N, x, y);
  // Launch it on one gpu thread like this:
  //add<<<1, 1>>>(N, x, y);
  add_threads<<<1, 255>>>(N, x, y);

  // And we need to wait for the gpu to finish
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  //delete [] x;
  //delete [] y;
  hipFree(x);
  hipFree(y);

  return 0;
}
